
#include <hip/hip_runtime.h>
#include <stdio.h>
#define N 10000

__global__ void add(int *a, int *b, int *c)		//tidak ada operasi di CPU. Ada 10 threads
{
	int tID= threadIdx.x;		//tID = selalu 1, namun blockID = menyesuaikan
	if (tID < N)
	{
		c[tID] = a[tID] + b[tID];		//blockID=0, tID=0, menjumlahkan a[0] dan b[0],
										//blockID=1, tID=0, menjumlahkan a[1] dan b[1],
										//blockID=2, tID=0, menjumlahkan a[2] dan b[2], dst.
	}
}

int main()
{
	int *a, *b, *c;
	a = (int*)malloc(N*sizeof(int));
	b = (int*)malloc(N*sizeof(int));
	c = (int*)malloc(N*sizeof(int));
	int *dev_a, *dev_b, *dev_c;
	
	hipMalloc((void **) &dev_a, N*sizeof(int));
	hipMalloc((void **) &dev_b, N*sizeof(int));
	hipMalloc((void **) &dev_c, N*sizeof(int));// Fill Arrays
	
	for (int i = 0; i < N; i++)
	{
		a[i] = i, b[i] = 1;		//CPU
	}
	
	hipMemcpy(dev_a, a, N*sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(dev_b, b, N*sizeof(int), hipMemcpyHostToDevice);
	add<<<1,N>>>(dev_a, dev_b, dev_c);
	hipMemcpy(c, dev_c, N*sizeof(int), hipMemcpyDeviceToHost);
	
	for (int i = 0; i < N; i++)
	{
		printf("%d + %d = %d\n", a[i], b[i], c[i]);
	}
	
	return 0;
}
